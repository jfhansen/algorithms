#include "hip/hip_runtime.h"
/*******************************************************************
 * File: mergesort.cu
 * Description: This file contains the CUDA implementation of
 * the MergeSort algorithm.
 * Slower than pure CPU implementation. Algorithm implemented both
 * using indices and using iterators.
 * For N = 2^20, the GPU takes 1.65s to run kernel function with 
 * index implementation, and 2.53s with iterator implementation.
 *
 * Author: jfhansen
 * Last Modification: 28/07/2020
 ******************************************************************/

#include <iostream>
#include <cstddef>
#include <assert.h>
#include <random>
#include <algorithm>
#include <string>
#include <limits>

#include "mergesort.hpp"

const int BLOCKSIZE = 256;

// Declare Kernel function, iterator implementation
// Arguments: src[], dst[], N, width, stride
__global__ void cudaMergeSortIterator(float*, float*, unsigned, unsigned, unsigned);

// Declare device function, iterator implementation
// Arguments: src[], dst[], first, mid, end
__device__ void cudaMergeIterator(float*, float*, float*, float*, float*);

// Declare Kernel function, index implementation
// Arguments: src[], dst[], N, width, stride
__global__ void cudaMergeSortIndexing(float*, float*, unsigned, unsigned, unsigned);

// Declare device function, index implementation
// Arguments: src[], dst[], first, mid, end
__device__ void cudaMergeIndex(float*, float*, unsigned, unsigned, unsigned);

// Returns global thread index
__device__ int getGlobalIdx();

// Performs merge sort with CUDA Kernel
// Arguments: data[], N, threads, blocks
void merge_sort(float*, unsigned, dim3, dim3);

int main(int argc, char *argv[]) {
	std::cout << "Beginning of main." << std::endl;

	size_t N;
	// If CLI argument not passed N = 2^20
	if (argc < 2)
		N = 1 << 20;
	// else fetch N from CLI argument
	else
		N = 1 << (std::stoi(argv[1]));

	std::cout << "N = " << N << std::endl;
	
	//N = 8;
	// Instantiate list on host
	float *h_list, *h_list_cpy;
	h_list = new float[N];
	h_list_cpy = new float[N];

	// Generate values from uniform distribution
	std::mt19937 rng;
	rng.seed(std::random_device()());
	std::uniform_real_distribution<float> dist(-100,100);

	// Fill host list
	std::generate(h_list, h_list+N, [&] { return dist(rng); });
	std::cout << "Generated list." << std::endl;
	
	//for (int i = 0; i < N; i++)
	//	h_list[i] = N-i;

	hipMemcpy(h_list_cpy, h_list, N*sizeof(float), hipMemcpyHostToHost);

	// Compute number of threads per block and total number of blocks
	unsigned blockSize = BLOCKSIZE;
	unsigned numBlocks = N/2/blockSize;

	dim3 threads(blockSize);
	dim3 blocks(numBlocks);

	merge_sort(h_list, N, threads, blocks);
	std::cout << "Finished GPU Merge sort." << std::endl;

	// Sort list on host device for comparison
	mergeSort(h_list_cpy, h_list_cpy+N);
	std::cout << "Finished CPU mergesort." << std::endl;

	// Compare sorted lists
	for (size_t i = 0; i < N; i++)
	{
		if (h_list[i] != h_list_cpy[i])
		{
			std::cout << "Element " << i << " does not match, host = " << h_list_cpy[i]
			<< ", device = " << h_list[i] << "." << std::endl;
			break;
		}
	}

	// Check for errors
	hipError_t err;
	while ( (err = hipGetLastError()) != hipSuccess )
		std::cout << "CUDA Error: " << hipGetErrorString(err) << std::endl;
	return 0;
}

void merge_sort(float *data, unsigned N, dim3 threads, dim3 blocks)
{
	// Allocate device memory
	float *d_data, *d_swap;
	hipMalloc(&d_data, N*sizeof(float));
	hipMalloc(&d_swap, N*sizeof(float));

	// Copy data to device
	hipMemcpy(d_data, data, N*sizeof(float), hipMemcpyHostToDevice);
	std::cout << "Copied data to device." << std::endl;
	
	float *src, *dst;
	src = d_data;
	dst = d_swap;

	// Calculate number of threads used as stride
	unsigned stride = blocks.x * blocks.y * blocks.z
		* threads.x * threads.y * threads.z;
	
	for (size_t width=2; width<(2*N); width*=2)
	{
		// Call kernel
		cudaMergeSortIterator<<<blocks, threads>>>(src, dst, N, width, stride);
		
		// Swap source and destination pointers for next iteration
		src = (src == d_data) ? d_swap : d_data;
		dst = (dst == d_data) ? d_swap : d_data;
	}
	
	// Copy sorted data to host memory.
	hipMemcpy(data, src, N*sizeof(float), hipMemcpyDeviceToHost);
}

__device__ int getGlobalIdx()
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x 
		+ blockIdx.z * gridDim.x * gridDim.y;
	int threadId = blockId * blockDim.x * blockDim.y * blockDim.z
		+ (threadIdx.z * blockDim.x * blockDim.y)
		+ threadIdx.y * blockDim.x 
		+ threadIdx.x;
	return threadId;
}

__device__ void cudaMergeIndex(float *src, float *dst, unsigned first, unsigned mid, unsigned last)
{
	unsigned i = first;
	unsigned j = mid;
	for (unsigned k = first; k < last; k++)
	{
		if (j >= last || i < mid && src[i] < src[j])
		{
			dst[k] = src[i];
			i++;
		}
		else
		{
			dst[k] = src[j];
			j++;
		}
	}
}

__global__ void cudaMergeSortIndexing(float *src, float *dst, unsigned N, unsigned width, unsigned stride)
{
	// Get global thread index
	unsigned tid = getGlobalIdx();
	
	unsigned first, mid, last;
	// Get index of first element in list that must be merged
	first = tid*width;

	for (size_t pair = tid; pair < (N/width); pair += stride)
	{
		if (first > N)
			break;
		// Get indices of middle element and past last element in list
		mid = min(first + width/2, N);
		last = min(first + width, N);

		// Call merge function on device
		cudaMergeIndex(src, dst, first, mid, last);
		
		first += width*stride;
	}
}

// Device function that merges left and right sublists
__device__ void cudaMergeIterator(float *src, float *dst, float *first, float *mid, float *last)
{
	// Get position of 'first' and 'last' in src array
	size_t pos_first, pos_last;
	pos_first = first - src;
	pos_last = last - src;

	// Iterators for dst array, left list and right list
	float *it, *it_l, *it_r;
	it_l = first;
	it_r = mid;
	
	for (it = (dst+pos_first); it < (dst+pos_last); it++)
	{
		if (it_r >= last || it_l < mid && *it_l < *it_r)
		{
			*it = *it_l;
			it_l++;
		}
		else
		{
			*it = *it_r;
			it_r++;
		}
	}
}

__global__ void cudaMergeSortIterator(float *src, float *dst, unsigned N, unsigned width, unsigned stride)
{
	// Get global thread index
	unsigned tid = getGlobalIdx();
	
	float *first, *mid, *last;
	// Get pointer to first element in list that must be merged
	first = (src + tid*width);

	for (size_t pair = tid; pair < (N/width); pair += stride)
	{
		if ((size_t)(first-src) > N)
			break;
		mid = (first+width/2 > src+N) ? src+N : first+width/2;
		last = (first+width > src+N) ? src+N : first+width;
		
		// Call merge function on device
		cudaMergeIterator(src, dst, first, mid, last);
		
		first += width*stride;
	}
}

